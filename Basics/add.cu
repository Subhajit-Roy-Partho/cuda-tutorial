
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add(int a, int b, int *c){
    *c = a + b;
};

int main(void){
    int c;
    int *c2;
    hipMalloc((void**)&c2,sizeof(int));
    add<<<1,1>>>(2,7,c2);
    hipMemcpy(&c,c2,sizeof(int),hipMemcpyDeviceToHost);
    cout<< "2+7 =" << c <<endl;
    hipFree(c2);

    return 0;
}