
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define N 1000

__global__ void add(int *a, int *b, int *c){
    int tid = blockIdx.x;
    if (tid<N)
        c[tid] = a[tid] + b[tid];
}

int main(void){
    int a[N],b[N],c[N];
    int *devA, *devB, *devC;

    hipMalloc((void**)&devA,N*sizeof(int));
    hipMalloc((void**)&devB,N*sizeof(int));
    hipMalloc((void**)&devC,N*sizeof(int));

    for(int i=0;i<N;i++){
        a[i]=-i;
        b[i]=i*i;
    }

    hipMemcpy(devA,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devB,b,N*sizeof(int),hipMemcpyHostToDevice);

    add<<<N,1>>>(devA,devB,devC);

    hipMemcpy(c,devC,N*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++) cout << a[i] << " + " << b[i] <<" = "<<c[i]<<endl;

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    return 0;
}